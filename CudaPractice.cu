
#include "hip/hip_runtime.h"


#include <stdio.h>
#include<iostream>
using namespace std;


// Hello world in CUDA world 

/*

__global__ void mykernel(void)
{
	printf("Hello this is GEFORECE GTX 1650 \n");
}
int main(void)
{
	mykernel <<<1,1>>> ();
	cudaDeviceSynchronize();
	printf("Hello this is Intel i5 10 Gen \n");
	return 0;
}


*/



// Add two numbers in cuda world


/*
__global__ void add(int* a, int* b, int* c)
{
	*c = *a + *b;
}

int main(void)
{
	int a, b, c;
	int* d_a, * d_b, * d_c;
	int size = sizeof(int);
	cudaMalloc((void**)&d_a, size);
	cudaMalloc((void**)&d_b, size);
	cudaMalloc((void**)&d_c, size);
	a = 2;
	b = 3;
	cudaMemcpy(d_a, &a, size, cudaMemcpyHostToDevice);
	cudaMemcpy(d_b, &b, size, cudaMemcpyHostToDevice);
	cudaMemcpy(d_c, &c, size, cudaMemcpyHostToDevice);
	add <<<1, 1 >>> (d_a, d_b, d_c);
	cudaDeviceSynchronize();
	cudaMemcpy(&c, d_c, size, cudaMemcpyDeviceToHost);
	printf("%d",c);
	cudaFree(d_a);
	cudaFree(d_b);
	cudaFree(d_c);
	return 0;
}


*/



// Vector Addition 


 
__global__ void add(int* a, int* b, int* c) {
	c[threadIdx.x] = a[threadIdx.x] / b[threadIdx.x];
}



void random_ints(int* a, int N)
{
	int i;
	for (i = 0; i < N; ++i)
		a[i] = rand();
}

#define N 10000
int main(void) {
	int* a, * b, * c;
	int* d_a, * d_b, * d_c;
	int size = N * sizeof(int);
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
	a = (int*)malloc(size);
	random_ints(a, N);
	b = (int*)malloc(size);
	random_ints(b, N);
	c = (int*)malloc(size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	add <<<1,N >>> (d_a, d_b, d_c);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	for (int i = 1 ; i < N; i++)
	{
		printf("%d\n", c[i] );
	}
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}





